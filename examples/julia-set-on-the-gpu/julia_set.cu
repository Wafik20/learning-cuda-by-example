#include "hip/hip_runtime.h"
#include <SDL2/SDL.h>
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

#define DIM_X 1920
#define DIM_Y 1080
uint32_t bitmap[DIM_X * DIM_Y];

// Complex number struct
struct hipComplex
{
    float r, i;
    __host__ __device__ hipComplex(float a, float b) : r(a), i(b) {}
    __host__ __device__ float magnitude2() const { return r * r + i * i; }
    __host__ __device__ hipComplex operator*(const hipComplex &a) const
    {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }
    __host__ __device__ hipComplex operator+(const hipComplex &a) const
    {
        return hipComplex(r + a.r, i + a.i);
    }
};

// Set a pixel in the bitmap (GPU-side)
__device__ void setPixel(uint32_t *bitmap, int x, int y, uint8_t R, uint8_t G, uint8_t B, uint8_t A)
{
    if (x >= 0 && x < DIM_X && y >= 0 && y < DIM_Y)
    {
        bitmap[y * DIM_X + x] = (A << 24) | (R << 16) | (G << 8) | B; // ARGB
    }
}

// Julia set calculation
__device__ int julia(int x, int y, float zoom)
{
    float scale = 1.5f / zoom;
    float jx = scale * (DIM_X / 2.0f - x) / (DIM_X / 2.0f);
    float jy = scale * (DIM_Y / 2.0f - y) / (DIM_Y / 2.0f);
    hipComplex c(-0.8f, 0.156f);
    hipComplex a(jx, jy);
    int i;
    for (i = 0; i < 200; i++)
    {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            break;
    }
    return i;
}


// CUDA kernel to compute the Julia set
__global__ void drawJuliaKernel(float zoom, uint32_t *bitmap)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < DIM_X && y < DIM_Y)
    {
        int iterations = julia(x, y, zoom);
        uint8_t gray = (iterations == 200) ? 0 : (uint8_t)(255.0f * iterations / 200.0f);
        setPixel(bitmap, x, y, gray, gray, gray, 255);
    }
}


int main()
{
    if (SDL_Init(SDL_INIT_VIDEO) != 0)
    {
        cerr << "SDL_Init Error: " << SDL_GetError() << endl;
        return 1;
    }

    SDL_Window *win = SDL_CreateWindow("Julia Set",
                                       SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED,
                                       DIM_X, DIM_Y, SDL_WINDOW_SHOWN);

    if (!win)
    {
        cerr << "SDL_CreateWindow Error: " << SDL_GetError() << endl;
        SDL_Quit();
        return 1;
    }

    SDL_Surface *screen = SDL_GetWindowSurface(win);

    SDL_Surface *image = SDL_CreateRGBSurfaceFrom(
        bitmap, DIM_X, DIM_Y, 32, DIM_X * sizeof(uint32_t),
        0x00FF0000, 0x0000FF00, 0x000000FF, 0xFF000000);

    if (!image)
    {
        cerr << "SDL_CreateRGBSurfaceFrom Error: " << SDL_GetError() << endl;
        SDL_DestroyWindow(win);
        SDL_Quit();
        return 1;
    }

    SDL_SetSurfaceBlendMode(image, SDL_BLENDMODE_NONE);

    float zoom = 1.0f;
    bool running = true;

    // Allocate GPU memory once, reuse
    uint32_t *d_bitmap;
    hipMalloc(&d_bitmap, DIM_X * DIM_Y * sizeof(uint32_t));

    while (running)
    {
        SDL_Event e;
        while (SDL_PollEvent(&e))
        {
            if (e.type == SDL_QUIT)
                running = false;
            if (e.type == SDL_KEYDOWN)
            {
                if (e.key.keysym.sym == SDLK_EQUALS || e.key.keysym.sym == SDLK_PLUS)
                    zoom *= 1.1f; // Zoom in
                if (e.key.keysym.sym == SDLK_MINUS)
                    zoom /= 1.1f; // Zoom out
            }
        }

        // Launch kernel
        dim3 blockSize(16, 16);
        dim3 gridSize((DIM_X + blockSize.x - 1) / blockSize.x,
                      (DIM_Y + blockSize.y - 1) / blockSize.y);
        drawJuliaKernel<<<gridSize, blockSize>>>(zoom, d_bitmap);
        hipDeviceSynchronize(); // Wait for GPU to finish

        // Copy result to host
        hipMemcpy(bitmap, d_bitmap, DIM_X * DIM_Y * sizeof(uint32_t), hipMemcpyDeviceToHost);

        // Update SDL surface
        SDL_LockSurface(image);
        SDL_UnlockSurface(image);
        SDL_BlitSurface(image, NULL, screen, NULL);
        SDL_UpdateWindowSurface(win);
    }

    hipFree(d_bitmap);
    SDL_FreeSurface(image);
    SDL_DestroyWindow(win);
    SDL_Quit();
    return 0;
}
